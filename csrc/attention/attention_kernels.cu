#include "hip/hip_runtime.h"
/*
 * Adapted from https://github.com/NVIDIA/FasterTransformer/blob/release/v5.3_tag/src/fastertransformer/kernels/decoder_masked_multihead_attention/decoder_masked_multihead_attention_template.hpp
 * Copyright (c) 2023, The vLLM team.
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include "attention_dtypes.h"
#include "attention_utils.cuh"

#include <algorithm>

#define WARP_SIZE 32
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

namespace vllm {

// Utility function for attention softmax.
template<int NUM_WARPS>
inline __device__ float block_sum(float* red_smem, float sum) {
  // Decompose the thread index into warp / lane.
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;

  // Compute the sum per warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
  }

  // Warp leaders store the data to shared memory.
  if (lane == 0) {
    red_smem[warp] = sum;
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The warps compute the final sums.
  if (lane < NUM_WARPS) {
    sum = red_smem[lane];
  }

  // Parallel reduction inside the warp.
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
  }

  // Broadcast to other threads.
  return __shfl_sync(uint32_t(-1), sum, 0);
}




// TODO(woosuk): Merge the last two dimensions of the grid.
// Grid: (num_heads, num_seqs, max_num_partitions).
template<
  typename scalar_t,
  int HEAD_SIZE,
  int BLOCK_SIZE,
  int NUM_THREADS,
  int PARTITION_SIZE = 0> // Zero means no partitioning.
__device__ void paged_attention_kernel_fused(
  float* __restrict__ exp_sums,           // [num_seqs, num_heads, max_num_partitions]
  float* __restrict__ max_logits,         // [num_seqs, num_heads, max_num_partitions]
  scalar_t* __restrict__ out,             // [num_seqs, num_heads, max_num_partitions, head_size]
  const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
  scalar_t* __restrict__ k_cache,   // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  scalar_t* __restrict__ v_cache,   // [num_blocks, num_kv_heads, head_size, block_size]
  const int* __restrict__ head_mapping,   // [num_heads]
  const float scale,
  const int* __restrict__ block_tables,   // [num_seqs, max_num_blocks_per_seq]
  const int* __restrict__ context_lens,   // [num_seqs]
  const int max_num_blocks_per_seq,
  const float* __restrict__ alibi_slopes, // [num_heads]
  const int q_stride,
  const int kv_block_stride,
  const int kv_head_stride,
  const scalar_t* __restrict__ key,           // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,         // [num_tokens, num_heads, head_size]
  const int64_t* __restrict__ slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size,
  const int x
  ) {
  // ****************** Stage 1 ******************** //
  // First update the key and value caches.
  // only one token of kv since single query attention

  const int64_t kvupdate_slot_idx = slot_mapping[0];
  if (kvupdate_slot_idx < 0) {
    // Padding token that should be ignored.
    return;
  }

  const int64_t kvupdate_block_idx = kvupdate_slot_idx / block_size;
  const int64_t kvupdate_block_offset = kvupdate_slot_idx % block_size;

  const int nnn = num_heads * head_size;
  // head_size = 128 here.
  int global_thd_idx = blockIdx.y * (gridDim.x * blockDim.x) + blockIdx.x * blockDim.x + threadIdx.x;
  int global_thd_num = gridDim.y * gridDim.x * blockDim.x;
  for (int i = global_thd_idx; i < nnn; i += global_thd_num) {
    const int64_t src_key_idx = 0 * key_stride + i;       // token_idx = 0
    const int64_t src_value_idx = 0 * value_stride + i;   // token_idx = 0

    const int kvupdate_head_idx = i / head_size;
    const int kvupdate_head_offset = i % head_size;
    const int kvupdate_x_idx = kvupdate_head_offset / x;
    const int kvupdate_x_offset = kvupdate_head_offset % x;

    const int64_t tgt_key_idx = kvupdate_block_idx * num_heads * (head_size / x) * block_size * x
                                + kvupdate_head_idx * (head_size / x) * block_size * x
                                + kvupdate_x_idx * block_size * x
                                + kvupdate_block_offset * x
                                + kvupdate_x_offset;
    const int64_t tgt_value_idx = kvupdate_block_idx * num_heads * head_size * block_size
                                  + kvupdate_head_idx * head_size * block_size
                                  + kvupdate_head_offset * block_size
                                  + kvupdate_block_offset;
    k_cache[tgt_key_idx] = key[src_key_idx];
    v_cache[tgt_value_idx] = value[src_value_idx];
  }
  __syncthreads();

  // ****************** Stage 2 ******************** //
  // Compute the single query attention.
  // printf("Calling!\n");
  const int seq_idx = blockIdx.y;
  const int partition_idx = blockIdx.z;
  const int max_num_partitions = gridDim.z;
  // There is no partition at least in v1.
  constexpr bool USE_PARTITIONING = PARTITION_SIZE > 0;
  const int context_len = context_lens[seq_idx];
  if (USE_PARTITIONING && partition_idx * PARTITION_SIZE >= context_len) {
    // No work to do. Terminate the thread block.
    return;
  }

  const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
  const int num_blocks_per_partition = USE_PARTITIONING ? PARTITION_SIZE / BLOCK_SIZE : num_context_blocks;

  // [start_block_idx, end_block_idx) is the range of blocks to process.
  const int start_block_idx = USE_PARTITIONING ? partition_idx * num_blocks_per_partition : 0;
  const int end_block_idx = MIN(start_block_idx + num_blocks_per_partition, num_context_blocks);
  const int num_blocks = end_block_idx - start_block_idx;

  // [start_token_idx, end_token_idx) is the range of tokens to process.
  const int start_token_idx = start_block_idx * BLOCK_SIZE;
  const int end_token_idx = MIN(start_token_idx + num_blocks * BLOCK_SIZE, context_len);
  // printf("BLOCK_SIZE: %d\n", BLOCK_SIZE);
  const int num_tokens = end_token_idx - start_token_idx;

  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  // WARP_SIZE = 32, BLOCK_SIZE = 16
  // 2 THREADS in 1 GROUP
  constexpr int NUM_THREAD_GROUPS = NUM_THREADS / THREAD_GROUP_SIZE; // Note: This assumes THREAD_GROUP_SIZE divides NUM_THREADS
  assert(NUM_THREADS % THREAD_GROUP_SIZE == 0);
  constexpr int NUM_TOKENS_PER_THREAD_GROUP = DIVIDE_ROUND_UP(BLOCK_SIZE, WARP_SIZE);
  // NUM_TOKENS_PER_THREAD_GROUP = 1
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int thread_idx = threadIdx.x;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int lane = thread_idx % WARP_SIZE;

  const int head_idx = blockIdx.x;
  // const int num_heads = gridDim.x;
  const int kv_head_idx = head_mapping[head_idx];
  const float alibi_slope = alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];

  // A vector type to store a part of a key or a query.
  // The vector size is configured in such a way that the threads in a thread group
  // fetch or compute 16 bytes at a time.
  // For example, if the size of a thread group is 4 and the data type is half,
  // then the vector size is 16 / (4 * sizeof(half)) == 2.
  constexpr int VEC_SIZE = MAX(16 / (THREAD_GROUP_SIZE * sizeof(scalar_t)), 1);
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;

  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;

  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;

  // Load the query to registers. (Only load one head of one query here. -> One block, one query head)
  // Each thread in a thread group has a different part of the query.
  // For example, if the the thread group size is 4, then the first thread in the group
  // has 0, 4, 8, ... th vectors of the query, and the second thread has 1, 5, 9, ...
  // th vectors of the query, and so on.
  // NOTE(woosuk): Because q is split from a qkv tensor, it may not be contiguous.
  const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
  __shared__ Q_vec q_vecs[THREAD_GROUP_SIZE][NUM_VECS_PER_THREAD];
  // q_vecs should have a capacity of HEAD_SIZE / VEC_SIZE, each element is of size VEC_SIZE
  // if (blockIdx.x + blockIdx.y + threadIdx.x == 0)
  // {
  //   printf("thread_group_idx: %d\n", thread_group_idx);
  //   printf("thread_group_offset: %d\n", thread_group_offset);
  //   printf("NUM_VECS_PER_THREAD: %d\n", NUM_VECS_PER_THREAD);
  //   printf("NUM_THREAD_GROUPS: %d\n", NUM_THREAD_GROUPS);
  // }
#pragma unroll
  for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD; i += NUM_THREAD_GROUPS) {
    // i = thread_group_idx; i < 16; i += 64 
    // Not actually a loop in our case.
    // Only load through the first several threads.
    const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
    // almost equal to thread_idx, unless it is iterated
    q_vecs[thread_group_offset][i] = *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
  }
  __syncthreads(); // TODO(naed90): possible speedup if this is replaced with a memory wall right before we use q_vecs

  // Memory planning.
  extern __shared__ char shared_mem[];
  // NOTE(woosuk): We use FP32 for the softmax logits for better accuracy.
  float* logits = reinterpret_cast<float*>(shared_mem);
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // x == THREAD_GROUP_SIZE * VEC_SIZE
  // Each thread group fetches x elements from the key at a time.
  // constexpr int x = 16 / sizeof(scalar_t);  // Can be derived from the definitions.
  float qk_max = -FLT_MAX;

  // Iterate over the key blocks.
  // Each warp fetches a block of keys for each iteration.
  // Each thread group in a warp fetches a key from the block, and computes
  // dot product with the query.
  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    // NOTE(woosuk): The block number is stored in int32. However, we cast it to int64
    // because int32 can lead to overflow when this variable is multiplied by large numbers
    // (e.g., kv_block_stride).
    const int64_t physical_block_number = static_cast<int64_t>(block_table[block_idx]);

    // Load a key to registers.
    // Each thread in a thread group has a different part of the key.
    // For example, if the the thread group size is 4, then the first thread in the group
    // has 0, 4, 8, ... th vectors of the key, and the second thread has 1, 5, 9, ... th
    // vectors of the key, and so on.
    for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
      // NUM_TOKENS_PER_THREAD_GROUP = 1
      // There is no actual iteration here.
      const int physical_block_offset = (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
      // thread_group_idx : 0-15, BLOCK_SIZE = 16
      const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      K_vec k_vecs[NUM_VECS_PER_THREAD];

#pragma unroll
      for (int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        const scalar_t* k_ptr = k_cache + physical_block_number * kv_block_stride
                                        + kv_head_idx * kv_head_stride
                                        + physical_block_offset * x;
        const int vec_idx = thread_group_offset + j * THREAD_GROUP_SIZE;
        // thread_group_offset = 0,1; THREAD_GROUP_SIZE = 2
        const int offset1 = (vec_idx * VEC_SIZE) / x;
        // Each THD_GROUP fetches x elements at a time
        // It is not determining group idx here. It is calculating the current round idx of current group.
        // x = 8
        const int offset2 = (vec_idx * VEC_SIZE) % x;
        k_vecs[j] = *reinterpret_cast<const K_vec*>(k_ptr + offset1 * BLOCK_SIZE * x + offset2);
      }

      // Compute dot product.
      // This includes a reduction across the threads in the same thread group.
      float qk = scale * Qk_dot<scalar_t, THREAD_GROUP_SIZE>::dot(q_vecs[thread_group_offset], k_vecs);
      // Add the ALiBi bias if slopes are given.
      qk += (alibi_slope != 0) ? alibi_slope * (token_idx - context_len + 1) : 0;

      if (thread_group_offset == 0) {
        // Store the partial reductions to shared memory.
        // NOTE(woosuk): It is required to zero out the masked logits.
        const bool mask = token_idx >= context_len;
        logits[token_idx - start_token_idx] = mask ? 0.f : qk;
        // Update the max value.
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }

  // Perform reduction across the threads in the same warp to get the
  // max qk value for each "warp" (not across the thread block yet).
  // The 0-th thread of each thread group already has its max qk value.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = qk_max;
  }
  __syncthreads();

  // TODO(woosuk): Refactor this part.
  // Get the max qk value for the sequence.
  qk_max = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }
  // Broadcast the max qk value to all threads.
  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  // Get the sum of the exp values.
  float exp_sum = 0.f;
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }
  exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], exp_sum);

  // Compute softmax.
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();

  // If partitioning is enabled, store the max logit and exp_sum.
  if (USE_PARTITIONING && thread_idx == 0) {
    float* max_logits_ptr = max_logits + seq_idx * num_heads * max_num_partitions
                                       + head_idx * max_num_partitions
                                       + partition_idx;
    *max_logits_ptr = qk_max;
    float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions
                                   + head_idx * max_num_partitions
                                   + partition_idx;
    *exp_sums_ptr = exp_sum;
  }

  // Each thread will fetch 16 bytes from the value cache at a time.
  constexpr int V_VEC_SIZE = MIN(16 / sizeof(scalar_t), BLOCK_SIZE);
  // printf("V_VEC_SIZE: %d\n", V_VEC_SIZE);
  using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using L_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using Float_L_vec = typename FloatVec<L_vec>::Type;

  constexpr int NUM_V_VECS_PER_ROW = BLOCK_SIZE / V_VEC_SIZE;
  constexpr int NUM_ROWS_PER_ITER = WARP_SIZE / NUM_V_VECS_PER_ROW;
  constexpr int NUM_ROWS_PER_THREAD = DIVIDE_ROUND_UP(HEAD_SIZE, NUM_ROWS_PER_ITER);

  // NOTE(woosuk): We use FP32 for the accumulator for better accuracy.
  float accs[NUM_ROWS_PER_THREAD];
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    accs[i] = 0.f;
  }
  // accs is for accumulating V vecs

  scalar_t zero_value;
  zero(zero_value);
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    // NOTE(woosuk): The block number is stored in int32. However, we cast it to int64
    // because int32 can lead to overflow when this variable is multiplied by large numbers
    // (e.g., kv_block_stride).
    const int64_t physical_block_number = static_cast<int64_t>(block_table[block_idx]);
    const int physical_block_offset = (lane % NUM_V_VECS_PER_ROW) * V_VEC_SIZE;
    const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
    L_vec logits_vec;
    from_float(logits_vec, *reinterpret_cast<Float_L_vec*>(logits + token_idx - start_token_idx));

    const scalar_t* v_ptr = v_cache + physical_block_number * kv_block_stride
                                    + kv_head_idx * kv_head_stride;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE) {
        const int offset = row_idx * BLOCK_SIZE + physical_block_offset;
        V_vec v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
        if (block_idx == num_context_blocks - 1) {
          // NOTE(woosuk): When v_vec contains the tokens that are out of the context,
          // we should explicitly zero out the values since they may contain NaNs.
          // See https://github.com/vllm-project/vllm/issues/641#issuecomment-1682544472
          scalar_t* v_vec_ptr = reinterpret_cast<scalar_t*>(&v_vec);
#pragma unroll
          for (int j = 0; j < V_VEC_SIZE; j++) {
            v_vec_ptr[j] = token_idx + j < context_len ? v_vec_ptr[j] : zero_value;
          }
        }
        accs[i] += dot(logits_vec, v_vec);
      }
    }
  }

  // Perform reduction within each warp.
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    float acc = accs[i];
#pragma unroll
    for (int mask = NUM_V_VECS_PER_ROW / 2; mask >= 1; mask /= 2) {
      acc += __shfl_xor_sync(uint32_t(-1), acc, mask);
    }
    accs[i] = acc;
  }

  // NOTE(woosuk): A barrier is required because the shared memory space for logits
  // is reused for the output.
  __syncthreads();

  // Perform reduction across warps.
  float* out_smem = reinterpret_cast<float*>(shared_mem);
#pragma unroll
  for (int i = NUM_WARPS; i > 1; i /= 2) {
    int mid = i / 2;
    // Upper warps write to shared memory.
    if (warp_idx >= mid && warp_idx < i) {
      float* dst = &out_smem[(warp_idx - mid) * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          dst[row_idx] = accs[i];
        }
      }
    }
    __syncthreads();

    // Lower warps update the output.
    if (warp_idx < mid) {
      const float* src = &out_smem[warp_idx * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          accs[i] += src[row_idx];
        }
      }
    }
    __syncthreads();
  }

  // Write the final output.
  if (warp_idx == 0) {
    scalar_t* out_ptr = out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE
                            + head_idx * max_num_partitions * HEAD_SIZE
                            + partition_idx * HEAD_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
        from_float(*(out_ptr + row_idx), accs[i]);
      }
    }
  }
}


// Grid: (num_heads, num_seqs, 1).
template<
  typename scalar_t,
  int HEAD_SIZE,
  int BLOCK_SIZE,
  int NUM_THREADS>
__global__ void paged_attention_v1_kernel(
  scalar_t* __restrict__ out,             // [num_seqs, num_heads, head_size]
  const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
  scalar_t* __restrict__ k_cache,         // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  scalar_t* __restrict__ v_cache,         // [num_blocks, num_kv_heads, head_size, block_size]
  const int* __restrict__ head_mapping,   // [num_heads]
  const float scale,
  const int* __restrict__ block_tables,   // [num_seqs, max_num_blocks_per_seq]
  const int* __restrict__ context_lens,   // [num_seqs]
  const int max_num_blocks_per_seq,
  const float* __restrict__ alibi_slopes, // [num_heads]
  const int q_stride,
  const int kv_block_stride,
  const int kv_head_stride,
  const scalar_t* __restrict__ key,           // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,         // [num_tokens, num_heads, head_size]
  const int64_t* __restrict__ slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size,
  const int x
) {
  paged_attention_kernel_fused<scalar_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>(
    /* exp_sums */ nullptr, /* max_logits */ nullptr,
    out, q, k_cache, v_cache, head_mapping, scale, block_tables, context_lens,
    max_num_blocks_per_seq, alibi_slopes, q_stride, kv_block_stride, kv_head_stride,
    key, value, slot_mapping, key_stride, value_stride, num_heads, head_size, block_size, x);
}




} // namespace vllm

#define LAUNCH_PAGED_ATTENTION_V1(HEAD_SIZE)                                                  \
  hipFuncSetAttribute(reinterpret_cast<const void*>(                                                                       \
    vllm::paged_attention_v1_kernel<T), HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>,                   \
    hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);                            \
  vllm::paged_attention_v1_kernel<T, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS>                      \
  <<<grid, block, shared_mem_size, stream>>>(                                                 \
    out_ptr,                                                                                  \
    query_ptr,                                                                                \
    key_cache_ptr,                                                                            \
    value_cache_ptr,                                                                          \
    head_mapping_ptr,                                                                         \
    scale,                                                                                    \
    block_tables_ptr,                                                                         \
    context_lens_ptr,                                                                         \
    max_num_blocks_per_seq,                                                                   \
    alibi_slopes_ptr,                                                                         \
    q_stride,                                                                                 \
    kv_block_stride,                                                                          \
    kv_head_stride,                                                                           \
    key_ptr,                                                                                  \
    value_ptr,                                                                                \
    slot_mapping_ptr,                                                                         \
    key_stride,                                                                               \
    value_stride,                                                                             \
    num_heads,                                                                                \
    head_size,                                                                                \
    block_size,                                                                               \
    x);


// TODO(woosuk): Tune NUM_THREADS.
template<
  typename T,
  int BLOCK_SIZE,
  int NUM_THREADS = 128>
void paged_attention_v1_launcher(
  torch::Tensor& out,
  torch::Tensor& query,
  torch::Tensor& key_cache,
  torch::Tensor& value_cache,
  torch::Tensor& head_mapping,
  float scale,
  torch::Tensor& block_tables,
  torch::Tensor& context_lens,
  int max_context_len,
  const c10::optional<torch::Tensor>& alibi_slopes,
  torch::Tensor& key,           // [num_tokens, num_heads, head_size]
  torch::Tensor& value,         // [num_tokens, num_heads, head_size]
  torch::Tensor& slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  // const int num_heads,
  // const int head_size,
  const int block_size,
  const int x
  ) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr = alibi_slopes ?
    reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
    : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  T* key_cache_ptr = reinterpret_cast<T*>(key_cache.data_ptr());
  T* value_cache_ptr = reinterpret_cast<T*>(value_cache.data_ptr());
  T* key_ptr = reinterpret_cast<T*>(key.data_ptr());
  T* value_ptr = reinterpret_cast<T*>(value.data_ptr());

  int* head_mapping_ptr = reinterpret_cast<int*>(head_mapping.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* context_lens_ptr = context_lens.data_ptr<int>();
  int64_t* slot_mapping_ptr = slot_mapping.data_ptr<int64_t>();

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int padded_max_context_len = DIVIDE_ROUND_UP(max_context_len, BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_context_len * sizeof(float);
  int outputs_size = (NUM_WARPS / 2) * head_size * sizeof(float);
  // Python-side check in vllm.worker.worker._check_if_can_support_max_seq_len
  // Keep that in sync with the logic here!
  int shared_mem_size = std::max(logits_size, outputs_size);

  dim3 grid(num_heads, num_seqs, 1);
  dim3 block(NUM_THREADS);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model. However, we can easily extend this
    // to support any head size which is a multiple of 16.
    case 64:
      LAUNCH_PAGED_ATTENTION_V1(64);
      break;
    case 80:
      LAUNCH_PAGED_ATTENTION_V1(80);
      break;
    case 96:
      LAUNCH_PAGED_ATTENTION_V1(96);
      break;
    case 112:
      LAUNCH_PAGED_ATTENTION_V1(112);
      break;
    case 128:
      LAUNCH_PAGED_ATTENTION_V1(128);
      break;
    case 256:
      LAUNCH_PAGED_ATTENTION_V1(256);
      break;
    default:
      TORCH_CHECK(false, "Unsupported head size: ", head_size);
      break;
  }
}

#define CALL_V1_LAUNCHER(T, BLOCK_SIZE)                             \
  paged_attention_v1_launcher<T, BLOCK_SIZE>(                       \
    out,                                                            \
    query,                                                          \
    key_cache,                                                      \
    value_cache,                                                    \
    head_mapping,                                                   \
    scale,                                                          \
    block_tables,                                                   \
    context_lens,                                                   \
    max_context_len,                                                \
    alibi_slopes,                                                   \
    key,                                                            \
    value,                                                          \
    slot_mapping,                                                   \
    key_stride,                                                     \
    value_stride,                                                   \
    block_size,                                                     \
    x);

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V1_LAUNCHER_BLOCK_SIZE(T)                              \
  switch (block_size) {                                             \
    case 8:                                                         \
      CALL_V1_LAUNCHER(T, 8);                                       \
      break;                                                        \
    case 16:                                                        \
      CALL_V1_LAUNCHER(T, 16);                                      \
      break;                                                        \
    case 32:                                                        \
      CALL_V1_LAUNCHER(T, 32);                                      \
      break;                                                        \
    default:                                                        \
      TORCH_CHECK(false, "Unsupported block size: ", block_size);   \
      break;                                                        \
  }

void paged_attention_v1(
  torch::Tensor& out,             // [num_seqs, num_heads, head_size]
  torch::Tensor& query,           // [num_seqs, num_heads, head_size]
  torch::Tensor& key_cache,       // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,     // [num_blocks, num_heads, head_size, block_size]
  torch::Tensor& head_mapping,    // [num_heads]
  float scale,
  torch::Tensor& block_tables,    // [num_seqs, max_num_blocks_per_seq]
  torch::Tensor& context_lens,    // [num_seqs]
  int block_size,
  int max_context_len,
  const c10::optional<torch::Tensor>& alibi_slopes,
  torch::Tensor& key,           // [num_tokens, num_heads, head_size]
  torch::Tensor& value,         // [num_tokens, num_heads, head_size]
  torch::Tensor& slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  // const int num_heads,
  // const int head_size,
  // const int block_size,
  const int x
  ) {
  if (query.dtype() == at::ScalarType::Float) {
    CALL_V1_LAUNCHER_BLOCK_SIZE(float);
  } else if (query.dtype() == at::ScalarType::Half) {
    CALL_V1_LAUNCHER_BLOCK_SIZE(uint16_t);
  } else if (query.dtype() == at::ScalarType::BFloat16) {
    CALL_V1_LAUNCHER_BLOCK_SIZE(__hip_bfloat16);
  } else {
    TORCH_CHECK(false, "Unsupported data type: ", query.dtype());
  }
}



#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
